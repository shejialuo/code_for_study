#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

void checkResult(float *hostRef, float *gpuRef, const int N) {
  double epsilon = 1.0E-8;
  bool match = 1;
  for (int i = 0; i < N; i++) {
    if (abs(hostRef[i] - gpuRef[i]) > epsilon) {
      match = 0;
      printf("Arrays do not match!\n");
      printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i], gpuRef[i], i);
      break;
    }
  }

  if (match) {
    printf("Arrays match.\n\n");
  }
}

void initialData(float *ip, int size) {
  time_t t;
  srand((unsigned)time(&t));

  for (int i = 0; i < size; i++) {
    ip[i] = (float)(rand() & 0xFF) / 10.0f;
  }
}

void sumMatrixOnHost(float *A, float *B, float *C, const int nx, const int ny) {
  float *ia = A;
  float *ib = B;
  float *ic = C;

  for (int iy = 0; iy < ny; iy++) {
    for (int ix = 0; ix < nx; ix++) {
      ic[ix] = ia[ix] + ib[ix];
    }
    ia += nx;
    ib += nx;
    ic += nx;
  }
}

__global__ void sumMatrixOnGPU2D(float *MatA, float *MatB, float *MatC, int nx,
                                 int ny) {
  unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
  unsigned int idx = iy * nx + ix;

  if (ix < nx && iy < ny) {
    MatC[idx] = MatA[idx] + MatB[idx];
  }
}

int main(int argc, char *argv[]) {
  printf("%s Starting...\n", argv[0]);

  int dev = 0;
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev);
  printf("Using Device %d : %s\n", dev, deviceProp.name);

  int nx = 1 << 14;
  int ny = 1 << 14;

  int nxy = nx * ny;
  int nBytes = nxy * sizeof(float);
  printf("Matrix size: nx %d ny %d\n", nx, ny);

  float *h_A, *h_B, *hostRef, *gpuRef;
  h_A = (float *)malloc(nBytes);
  h_B = (float *)malloc(nBytes);
  hostRef = (float *)malloc(nBytes);
  gpuRef = (float *)malloc(nBytes);

  initialData(h_A, nxy);
  initialData(h_B, nxy);

  memset(hostRef, 0, nBytes);
  memset(gpuRef, 0, nBytes);

  sumMatrixOnHost(h_A, h_B, hostRef, nx, ny);

  float *d_MatA, *d_MatB, *d_MatC;
  hipMalloc((void **)&d_MatA, nBytes);
  hipMalloc((void **)&d_MatB, nBytes);
  hipMalloc((void **)&d_MatC, nBytes);

  hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice);

  int dimx = 32;
  int dimy = 32;
  dim3 block(dimx, dimy);
  dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

  sumMatrixOnGPU2D<<<grid, block>>>(d_MatA, d_MatB, d_MatC, nx, ny);
  hipDeviceSynchronize();

  hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost);

  checkResult(hostRef, gpuRef, nxy);

  hipFree(d_MatA);
  hipFree(d_MatB);
  hipFree(d_MatC);

  free(h_A);
  free(h_B);
  free(hostRef);
  free(gpuRef);

  hipDeviceReset();
  return 0;
}
