#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void nestedHelloWorld(const int iSize, int iDepth) {
  int tid = threadIdx.x;

  printf("Recursion=%d: Hello World from thread %d block %d\n", iDepth, tid,
         blockIdx.x);

  if (iSize == 1) {
    return;
  }

  int nthreads = iSize >> 1;
  if (tid == 0 && nthreads > 0) {
    nestedHelloWorld<<<1, nthreads>>>(nthreads, ++iDepth);
    printf("-------> nested execution depth: %d\n", iDepth);
  }
}

int main(int argc, char *argv[]) {
  int size = 8;
  int blocksize = 8;
  int igrid = 1;
  if (argc > 1) {
    igrid = atoi(argv[1]);
    size = igrid * blocksize;
  }

  dim3 block(blocksize, 1);
  dim3 grid((size + block.x - 1) / block.x,  1);
  printf("%s Execution Configuration: grid %d block %d\n", argv[0], grid.x, block.x);

  nestedHelloWorld<<<grid, block>>>(block.x, 0);
  hipDeviceSynchronize();

  return 0;
}